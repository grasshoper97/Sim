//#include "cuda_runtime.h"
//#include <cuda.h>
//#include <cuda_runtime_api.h>
//#include "device_launch_parameters.h"

#include <hip/hip_runtime.h>
#include<stdio.h>
//#include<stdlib.h>
//#include<string.h>
//#include<math.h>
//#include<cutil.h>
 

#include<iostream>
#define NUM 2048

//////////////////////////////////////////////////////////////////
__global__ void 
    caculate(unsigned char * g_x)
{
    __shared__ char s[512];
	const unsigned int bid=blockIdx.x*blockDim.x+threadIdx.x;
    if(bid>=NUM) return;
    s[bid]=g_x[bid];
     s[bid]*=2;  
     g_x[bid]=s[bid];

}
 
///////////////////////////////////////////////////////////////
    int main(int argc,char**argv)
{


     //重定向到文件
    //freopen("1.txt", "w", stdout);
    int SIZE=sizeof(unsigned char);
    //----------------------------------------
    unsigned char *h_x=(unsigned char*)malloc(SIZE*NUM);
    for(int i=0;i<NUM;i++)
        h_x[i]=100;  
    //---------------------------
    unsigned char *d_x;    
    hipMalloc((void**)&d_x,SIZE*NUM);

    //输入数组从内存拷贝到显存
    hipMemcpy(d_x,h_x,SIZE*NUM,hipMemcpyHostToDevice);

     //调用核心函数
    dim3 grid;
    dim3 block;
    block.x=512;
    grid.x=(NUM+block.x-1)/block.x;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    double sum=0;

    for(int i=0;i<1;i++){
    hipEventRecord(start, 0);
    float runTime;
    //====================================
    caculate<<<grid,block>>>(d_x);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);
    printf("kernel error no =[%d]",hipGetLastError()); 
    printf("time= %f\n ",runTime);
    sum+=runTime;

    }
    printf("aver time= %f\n ",sum);

    //两个同步语句cudaThreadSynchronize、cudaDeviceSynchronize必须有一个才能让nsight中显示内核函数。
    //cudaThreadSynchronize(); 
    //=====================================



    //CUT_CHECK_ERROR("Kernel execution failed");
    //输出数组从显存拷贝到内存
    hipMemcpy(h_x,d_x,SIZE*NUM,hipMemcpyDeviceToHost);
    //在主机端打印
    //for(int i=0;i<NUM;i++)
    printf("h_x[0]=[%c]\n",h_x[0]); 
    printf("h_x[num-1]=[%c]\n",h_x[NUM-1]); 

    //释放内存、显存    
    free(h_x);
    hipFree(d_x);


    printf("press enter to quit:");
    getchar();
}


